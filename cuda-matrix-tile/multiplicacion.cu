
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 70


__global__
void mult_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;


	if ( col<n && row<n )
	{
		int i;
		c[row*n+col] = 0;

		for(i=0;i<n;i++)
		{
			c[row*n + col] += a[ row*n + i ]*b[ i*n + col ];

		}

	}
}

__global__
void mult_matrix_shared(int* a, int* b, int* c,int n)
{
	__shared__ float Mds[THREAD_PER_BLOCK][THREAD_PER_BLOCK];
	__shared__ float Nds[THREAD_PER_BLOCK][THREAD_PER_BLOCK];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int Row = by * THREAD_PER_BLOCK + ty;
	int Col = bx * THREAD_PER_BLOCK + tx;

	int Pvalue = 0;
	
	for (int ph = 0; ph < n/THREAD_PER_BLOCK; ++ph) {
	
		Mds[ty][tx] = a[Row*n + ph*THREAD_PER_BLOCK + tx];
		Nds[ty][tx] = b[(ph*THREAD_PER_BLOCK + ty)*n + Col];
		__syncthreads();
		
		for (int k = 0; k < THREAD_PER_BLOCK; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	c[Row*n + Col] = Pvalue;
}


void fill_mat(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			a[i*n+j] = rand()%5+1;
		}
	}
}

int main()
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;

	int mat_elem = 2000;
	int my_size = mat_elem*mat_elem*sizeof(int);
	
	float tiempo;
	hipEvent_t inicio,final;
	hipEventCreate(&inicio);
	hipEventCreate(&final);

	a = (int*) malloc(my_size);
	b = (int*) malloc(my_size);
	c = (int*) malloc(my_size);

	fill_mat(a,mat_elem);
	fill_mat(b,mat_elem);
	printf("\n");

	hipMalloc((void**)&d_a,my_size);
	hipMalloc((void**)&d_b,my_size);
	hipMalloc((void**)&d_c,my_size);

	hipMemcpy(d_a,a,my_size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,my_size,hipMemcpyHostToDevice);

	dim3 my_block(THREAD_PER_BLOCK,THREAD_PER_BLOCK);
	dim3 my_grid((mat_elem + THREAD_PER_BLOCK-1)/my_block.x,(mat_elem + THREAD_PER_BLOCK-1)/my_block.y);

	
    	hipEventRecord(inicio,0);
	
	//mult_matrix_shared<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
	mult_matrix<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
    	hipEventRecord(final,0);
    	hipEventSynchronize(final);
    	/////////////////////////////////////////////////////

    	hipEventElapsedTime(&tiempo,inicio,final);

	hipMemcpy(c,d_c,my_size,hipMemcpyDeviceToHost);

	printf("tiempo %d X %d, tam=%d : %0.15f\n",THREAD_PER_BLOCK,THREAD_PER_BLOCK,mat_elem,tiempo);
	return 0;
}

