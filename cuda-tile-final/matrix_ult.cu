
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 2

__global__
void add_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;

	if ( col<n && row<n )
	{
		c[row*n+col] = a[row*n+col] + b[row*n+col];
	}
}

__global__
void mult_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;


	if ( col<n && row<n )
	{
		int i;
		c[row*n+col] = 0;

		for(i=0;i<n;i++)
		{
			c[row*n + col] += a[ row*n + i ]*b[ i*n + col ];

		}

	}
}

__global__
void mult_matrix_shared(int* a, int* b, int* c,int n)
{
	__shared__ float sub_a[THREAD_PER_BLOCK][THREAD_PER_BLOCK];
	__shared__ float sub_b[THREAD_PER_BLOCK][THREAD_PER_BLOCK];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int Row = by * THREAD_PER_BLOCK + ty;
	int Col = bx * THREAD_PER_BLOCK + tx;

	int Pvalue = 0;
	
	for (int ph = 0; ph < n/THREAD_PER_BLOCK; ++ph) {
	
		sub_a[ty][tx] = a[Row*n + ph*THREAD_PER_BLOCK + tx];
		sub_b[ty][tx] = b[(ph*THREAD_PER_BLOCK + ty)*n + Col];
		__syncthreads();
		
		for (int k = 0; k < THREAD_PER_BLOCK; ++k) {
			Pvalue += sub_a[ty][k] * sub_b[k][tx];
		}
		__syncthreads();
	}
	c[Row*n + Col] = Pvalue;
}

__global__ 
void  mult_mat_rectangular(int *d_M, int *d_N, int *p,int N){
	__shared__ int Mds[THREAD_PER_BLOCK][THREAD_PER_BLOCK];
	__shared__ int Nds[THREAD_PER_BLOCK][THREAD_PER_BLOCK];

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Row = by*THREAD_PER_BLOCK + ty;
	int Col = bx*2*THREAD_PER_BLOCK + tx;
	
	int Col2 = (bx*2 + 1)*THREAD_PER_BLOCK + tx;

	int p1 = 0;
	int p2 = 0;
	
	int k = 0;
	int prefM = d_M[Row*N + k*THREAD_PER_BLOCK + tx];
	int prefN = d_N[(k*THREAD_PER_BLOCK + ty)*N + Col];
	
	int prefN2 = d_N[(k*THREAD_PER_BLOCK + ty)*N + Col2];
		
	Mds[ty][tx] = prefM;
	Nds[ty][tx] = prefN;
	__syncthreads();
	
	for(int m = 0; m < N/THREAD_PER_BLOCK ; ++m){				
		
		prefM = d_M[Row*N + m*THREAD_PER_BLOCK + tx];
		prefN = d_N[(m*THREAD_PER_BLOCK + ty)*N + Col];
		
		for(int k = 0; k < THREAD_PER_BLOCK; k++){
			p1 += Mds[ty][k] * Nds[k][tx];
		}		
		
		__syncthreads();
		
		Nds[ty][tx] = prefN2;
		
		__syncthreads();
		
		prefN2 = d_N[(m*THREAD_PER_BLOCK + ty)*N + Col2];
		
		for(int k = 0; k < THREAD_PER_BLOCK; k++){
			p2 += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
		
		
		Mds[ty][tx] = prefM;
		Nds[ty][tx] = prefN;
		
	}
	p[Row*N + Col] = p1;
	p[Row*N + Col2] = p2;
}

void print_matrix(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			printf("%d ",a[i*n+j]);
		}
		printf("\n");
	}
}

void fill_matrix(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			//a[i*n+j] = rand()%5+1;
			a[i*n+j] = 1;
		}
	}
}

int main()
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;

	int mat_elem = 8;
	int my_size = mat_elem*mat_elem*sizeof(int);

	//cudaEvent_t my_start,my_stop;
	//cudaEventCreate(&my_start);
	//cudaEventCreate(&my_stop);

	a = (int*) malloc(my_size);
	b = (int*) malloc(my_size);
	c = (int*) malloc(my_size);

	fill_matrix(a,mat_elem);
	fill_matrix(b,mat_elem);

	printf("Matrix A\n");
	print_matrix(a,mat_elem);
	printf("Matrix B\n");
	print_matrix(b,mat_elem);
	printf("\n");

	hipMalloc((void**)&d_a,my_size);
	hipMalloc((void**)&d_b,my_size);
	hipMalloc((void**)&d_c,my_size);

	hipMemcpy(d_a,a,my_size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,my_size,hipMemcpyHostToDevice);

	dim3 my_block(THREAD_PER_BLOCK,THREAD_PER_BLOCK);
	dim3 my_grid((mat_elem + THREAD_PER_BLOCK-1)/my_block.x,(mat_elem + THREAD_PER_BLOCK-1)/my_block.y);
	
	//////////////////////ELAPSED TIME ///////////////////////////////
	
	//cudaEventRecord(my_start,0);
	//mult_matrix<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
	mult_mat_rectangular<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
	//cudaEventRecord(my_stop,0);
	//cudaEventSynchronize(my_stop);
	/////////////////////////////////////////////////////
	
	//float elapsed_time;
	//cudaEventElapsedTime(&elapsed_time,my_start,my_stop);

	hipMemcpy(c,d_c,my_size,hipMemcpyDeviceToHost);
	printf("Matrix C\n");
	print_matrix(c,mat_elem);	


	//printf("time : %f\n",elapsed_time);
	return 0;
}